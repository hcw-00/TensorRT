#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "myTestKernel.h"

template <unsigned nthdsPerCTA>
__launch_bounds__(nthdsPerCTA) __global__
    void myTestKernel(const int n, const float coefficient, const float* input, float* output)
{
    for (int i = blockIdx.x * nthdsPerCTA + threadIdx.x; i < n; i += gridDim.x * nthdsPerCTA)
    {
        // output[i] = input[i] > 0 ? input[i] : input[i] * negativeSlope;
        output[i] = input[i] + coefficient;
    }
}

pluginStatus_t myTestGPU(hipStream_t stream, const int n, const float coefficient, const void* input, void* output)
{
    const int BS = 512;
    const int GS = (n + BS - 1) / BS;
    myTestKernel<BS><<<GS, BS, 0, stream>>>(n, coefficient,
                                           (const float*) input,
                                           (float*) output);
    return STATUS_SUCCESS;
}

pluginStatus_t myTestInference(
    hipStream_t stream, const int n, const float coefficient, const void* input, void* output)
{
    return myTestGPU(stream, n, coefficient, (const float*) input, (float*) output);
}
